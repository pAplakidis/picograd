
#include <hip/hip_runtime.h>
extern "C" __global__ void relu_kernel(float *input, float *output, int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
  {
    output[idx] = fmaxf(0.0f, input[idx]);
  }
}